
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cassert>
#include <hiprand/hiprand.h>
#include <omp.h>
#include <cmath>
#include <time.h>

using namespace std;

typedef struct {
    int iter;
    int node;
    int init_infected_node;
    int map_width;
    int map_height;
    double max_moving_radius;
    double max_infection_radius;
    double alpha_constant;
    double beta_constant;
} Parameter;
Parameter param;

void extractParam(Parameter * param, char *argv[]) {
    param->iter = atoi(argv[1]);
    param->node = atoi(argv[2]);
    param->init_infected_node = atoi(argv[3]);
    param->map_width = atoi(argv[4]);
    param->map_height = atoi(argv[5]);
    param->max_moving_radius = atof(argv[6]);
    param->max_infection_radius = atof(argv[7]);
    param->alpha_constant = atof(argv[8]);
    param->beta_constant = atof(argv[9]);

    // Self-Checking
    cout << "*---------------------------Input Parameter---------------------------*" << endl << endl;
    cout << left << setw(60) << "# Number of Iteration: " << param->iter << endl;
    cout << left << setw(60) << "# Number of Nodes: " << param->node << endl;
    cout << left << setw(60) << "# Initial Number of Infectious Node: " << param->init_infected_node << endl;
    cout << left << setw(60) << "# Map Width: " << param->map_width << endl;
    cout << left << setw(60) << "# Map Height: " << param->map_height << endl;
    cout << left << setw(60) << "# Max Moving Radius: " << param->max_moving_radius << endl;
    cout << left << setw(60) << "# Max Infection Radius: " << param->max_infection_radius << endl;
    cout << left << setw(60) << "# Infection Ratio Alpha for Probability of Infection: " << param->alpha_constant << endl;
    cout << left << setw(60) << "# Infection Ratio Beta for Probability of Infection: " << param->beta_constant << endl;
    cout << endl << "*---------------------------------------------------------------------*" << endl;
}

// State Definition
#define NODE_STATE_SUSCEPTIBLE 0
#define NODE_STATE_INFECTIOUS 1
#define NODE_STATE_RECOVERED 2
#define NODE_STATE_DEAD 3

// Definition
#define NODE_MAX_VELOCITY 4

// Constant Definition
__constant__ float DeadProbability[22] = {0.0, 0.001, 0.002, 0.004, 0.008, 0.014, 0.022, 0.032, 0.022, 0.014, 0.008, 0.004, 0.002, 0.001, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
__constant__ float RecoveryRate[22] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.31, 0.43, 0.57, 0.73, 0.91, 1.0};

// Kernel
__global__ void computeDist(float * ddist, float * dmapX, float * dmapY, int nodeCount, float alpha, float beta) {
    // 用輪盤式的 Access 看會不會好一點
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= nodeCount || y >= nodeCount) return;

    // 計算 Dist
    float dist = sqrtf(powf(dmapX[x] - dmapX[y], 2) + powf(dmapY[x] - dmapY[y], 2));
    
    // Calculate Infactious Rate
    ddist[x*nodeCount+y] = alpha * expf(-1 * dist * beta);
}

// Probability 算數
inline double infectionRate(double dist) {
    // 如果拿 alpha = 0.99(最大值) beta = 2.5 distance = 3 --> 接近 0
    return param.alpha_constant * exp(-1 * dist * param.beta_constant);
}

double calculateTime(timespec * start, timespec * end) {
  timespec temp;
  if ((end->tv_nsec - start->tv_nsec) < 0) {
       temp.tv_sec = end->tv_sec-start->tv_sec-1;
       temp.tv_nsec = 1000000000 + end->tv_nsec - start->tv_nsec;
   } else {
       temp.tv_sec = end->tv_sec - start->tv_sec;
       temp.tv_nsec = end->tv_nsec - start->tv_nsec;
   }
   return temp.tv_sec + (double) temp.tv_nsec / 1000000000.0;
}

/*
    argv[1] --> Number of Iteration
    argv[2] --> Number of Nodes
    argv[3] --> Initial Number of Infectious Node
    argv[4] --> Map Width
    argv[5] --> Map Height
    argv[6] --> Max Moving Radius
    argv[7] --> Max Infection Radius
    argv[8] --> Infection Ratio Alpha for Probability of Infection
    argv[9] --> Infection Ratio Beta for Probability of Infection
*/
int main(int argc, char *argv[]) {

    assert(argc == 10);

    // Init Random Generator with a Seed
    srand(time(NULL));

    // Extract all Parameter to a Structure
    extractParam(&param, argv);

    timespec start, end;

    clock_gettime(CLOCK_MONOTONIC, &start);

    // 這邊考量到 Coleased Memory Access 的問題，把 X, Y 的位置分開儲存
    // 這裏可以寫兩個版本，一個是由 CPU 同時處理 Rand 的計算，另一是使用 curand
    // 把資料直接產生在 GPU 上
    float * mapX = reinterpret_cast<float *>(new float[param.node]);
    float * mapY = reinterpret_cast<float *>(new float[param.node]);
    float * dmapX, * dmapY;
    hipMalloc(&dmapX, param.node * sizeof(float));
    hipMalloc(&dmapY, param.node * sizeof(float));
    
    // State
    // dState 是放在 GPU 上的，理論上並不需要傳回來，但為了把資料輸出視覺化呈現
    // 可以在每一輪結束時把資料 Transfer 回來
    // unsigned short * dState;
    short * state = reinterpret_cast<short *>(new short[param.node]);
    short * nextState = reinterpret_cast<short *>(new short[param.node]);
    for (int i = 0; i < param.node; i++) state[i] = nextState[i] = NODE_STATE_SUSCEPTIBLE;

    int tmp, counter = param.init_infected_node;
    while (counter > 0) {
        tmp = rand() % param.node;
        if (state[tmp] != NODE_STATE_INFECTIOUS) {
            state[tmp] = nextState[tmp] = NODE_STATE_INFECTIOUS;
            counter--;
            // cout << "[Map::generate_node]: Node " << node_list[tmp].index << " Set Infected." << endl;
        }
    }

    // 我覺得 GPU 單純拿來計算 Probability 就好，同一時間 CPU 負責計算 Move 的結果
    float * prob = reinterpret_cast<float *>(new float[param.node]);

    // Initial Rand Position
    // 這裡試過了，千萬不要放 Openmp 的 code 來跑，效果有夠慢
    for (int i = 0; i < param.node; i++) {
        mapX[i] = rand() % (param.map_width * 1000) / 1000.0;
        mapY[i] = rand() % (param.map_height * 1000) / 1000.0;
    }

    // Create Distance Matrix
    float * dist = reinterpret_cast<float *>(new float[param.node * param.node]);
    float * ddist;
    hipMalloc(&ddist, param.node * param.node * sizeof(float));

    // Algorithm Main Part
    // GPU 每一有以下幾個 Step
    // 1. GPU Calculate Distance
    // 2. Calculate Probability
    // CPU 有以下幾個工作
    // 1. MOVE
    // 2. Change State
    const dim3 blocks(ceil(param.node/32.0), ceil(param.node/32.0));
    const dim3 threadBlock(32, 32);
    float moving[2];
    for (int iter = 0; iter < param.iter; iter++) {
        // 1. Async 搬移資料到 GPU 去
        // [TODO]: 不知道為什麼什麼都沒有過去
        hipMemcpyAsync(dmapX, mapX, param.node*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(dmapY, mapY, param.node*sizeof(float), hipMemcpyHostToDevice);

        // 2. Submit Kernel Function
        computeDist<<< blocks, threadBlock >>>(ddist, dmapX, dmapY, param.node, param.alpha_constant, param.beta_constant);

        // 3. CPU 同時計算 Move
        // [TODO] 這邊可以 Parallel 化
        for (int i = 0; i < param.node; i++) {
            moving[0] = mapX[i] += rand() % (NODE_MAX_VELOCITY * 2 * 1000) / 1000.0 - NODE_MAX_VELOCITY;
            moving[1] = mapY[i] += rand() % (NODE_MAX_VELOCITY * 2 * 1000) / 1000.0 - NODE_MAX_VELOCITY;
            if (moving[0] >= 0 && moving[0] <= param.map_width) mapX[i] = moving[0];
            if (moving[1] >= 0 && moving[1] <= param.map_height) mapY[i] = moving[1];
            state[i] = nextState[i];
        }

        // 4. 把資料搬回來
        hipMemcpy(dist, ddist, param.node * param.node *sizeof(float), hipMemcpyDeviceToHost);

        // 從 Dist 拿回來後，開始計算
        // [TODO]: 這邊可以用 openmp 來處理
#pragma omp parallel num_threads(16)
{
        #pragma omp for
        for (int n = 0; n < param.node; n++) {
            if (state[n] == NODE_STATE_RECOVERED || state[n] == NODE_STATE_DEAD) continue;
            float prob = 1;
            for (int k = 0; k < param.node; k++) {
                if (n == k || state[k] != NODE_STATE_INFECTIOUS) continue;

                if (dist[n*param.node + k] >= 0.1) prob *= dist[n*param.node + k];
            }
            if (prob == 1) prob = 0;

            // 5. State 計算
            if (prob >= 0.1) nextState[n] = NODE_STATE_INFECTIOUS;
        }
}
        cout << "Iteration: " << iter << " has completed." << endl;
    }

    clock_gettime(CLOCK_MONOTONIC, &end);
    std::cout << "[0]Output Time: " << calculateTime(&start, &end) << "(sec)" << std::endl;
    // Free Cuda Resource
    hipFree(dmapX);
    hipFree(dmapY);
    hipFree(ddist);

    // Free CPU Resources
    delete [] mapX;
    delete [] mapY;
    delete [] state;
    delete [] prob;
    delete [] dist;

    return 0;
}